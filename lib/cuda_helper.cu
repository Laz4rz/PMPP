#include "../include/cuda_helper.h"

void cudaMallocGuard( void **devPtr, size_t size ) {
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
